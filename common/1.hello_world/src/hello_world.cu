#include <hip/hip_runtime.h> /* core lib */

#include <stdio.h>

/* Device code - Kernel */
__global__ void hello_cuda()
{
    printf("Code execution on GPU\n");
}

/* Host code */
int
main(int argc, char** argv)
{
    printf("Code execution on CPU\n");

    /* The int param referes the number of threads runs on GPU */
    hello_cuda<<< 1, 1 >>> ();

    /* Host waits untils GPU Code returns */
    hipDeviceSynchronize();

    /* Destroys the primary context of the device the host
     * thread currently operates 
     */ 
    hipDeviceReset();
    return 0;
}