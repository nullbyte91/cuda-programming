#include <hip/hip_runtime.h> /* core lib */

#include <stdio.h>

/* Device code - Kernel */
__global__ void hello_cuda()
{
    printf("Code execution on GPU\n");
}

/* Host code */
int
main(int argc, char** argv)
{
    printf("Code execution on CPU\n");

    /* 32 threds in parallel
    *  8 grid each runs 4 block threads in x direction
    */

    dim3 grid_x(8, 1, 1);
    dim3 block_y(4, 1, 1);

    /* Grid, block */
    hello_cuda<<< grid_x, block_y >>> ();

    /* Host waits untils GPU Code returns */
    hipDeviceSynchronize();

    /* 32 threds in parallel
    *  8 grid in x, y direction each runs 4 block threads in x, y direction
    */

    dim3 grid_xy(8, 2, 1);
    dim3 block_xy(4, 1, 1);

    /* Grid, block */
    hello_cuda<<< grid_xy, block_xy >>> ();
    
    
    /* Host waits untils GPU Code returns */
    hipDeviceSynchronize();

    /* Destroys the primary context of the device the host
     * thread currently operates 
     */ 
    hipDeviceReset();
    return 0;
}